#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

using namespace std;

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define NUM_THREADS 1024
hipStream_t stream;

float ** training_x; //10000*1024 --> 2556*1024
float ** training_y; //10000*1 --> 2556*1
float ** testing_x; //504*1024
float ** testing_y; //504*1

void getData(float * res, char buff[])
{
    char *token = strtok(buff," ,");
    int counter=0;
    
    while( token != NULL )
    {
        counter++;
        res[counter-1] = atof(token);
        token = strtok(NULL," ,");
    }
}

void readCSV(char* file, float** mat, int x_dim, int y_dim)
{
    FILE* stream = fopen(file, "r");
    int size_per_pic = y_dim * 30;
    char line[size_per_pic];
    int num;
    if (stream == NULL) {
        perror ("Error opening file");
        return;
    }

    int i = 0;
    while (fgets(line, size_per_pic, stream))
    {
        char* tmp = strdup(line);
        getData(mat[i], tmp);
        i++;
    }
}

void malloc_host(void){
    training_x = (float**)malloc(sizeof(float*) * 10000);
    for(int i = 0; i < 2556; i++){
        training_x[i] = (float*)malloc(sizeof(float) * 1024);
    }

    training_y = (float**)malloc(sizeof(float*) * 10000);
    for(int i = 0; i < 2556; i++){
        training_y[i] = (float*)malloc(sizeof(float) * 1);
    }

    testing_x = (float **)malloc(sizeof(float*) * 2000);
    for(int i = 0; i < 504; i++){
        testing_x[i] = (float*)malloc(sizeof(float) * 1024);
    }

    testing_y = (float **)malloc(sizeof(float*) * 2000);
    for(int i = 0; i < 504; i++){
        testing_y[i] = (float*)malloc(sizeof(float) * 1);
    }
}

// void malloc_weight(void){
//     weight = (float*)malloc(sizeof(float*) * 1024);
    
// }

void initialize_weight(float* weight){
    for (int j = 0; j < 1024; j++) {
        weight[j] = 0;
        // printf("weight %f\n", weight[i][j]);
    }
}

// train_data size a[M][N]    M = data_size; N = 784;
// weight size b[N][S]        N = 784; S = 10
// result size result[M][S]   M = data_size; S = 10
void Mult_CPU( float * a, float * b, float *result,const int M,const int N,const int S) // M should be batch size
{
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < S; j++)
        {
            int index = i * S + j;
            result[index] = 0;

            //循环计算每一个元素的结果
            for (int k = 0; k < N; k++)
            {
                result[index] += a[i * N + k] * b[k * S + j];
            }
        }
    }
}

__global__ void Mult_GPU( float *a,  float *b, float *result, const int M, const int N, const int S) // M should be batch size
{
    int threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
                    + blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId < M * S)
    {
        int row = threadId / S;
        int column = threadId % S;

        result[threadId] = 0;
        for (int i = 0; i < N; i++)
        {
            result[threadId] += a[row * N + i] * b[i * S + column];
        }
    }
}

void sigmoid_CPU(float *a, float *result, const int size){
    for (int i = 0; i < size; i++){
        if(a[i] > 10){
            result[i] = 1;
        }
        else if(a[i] < -10){
            result[i] = 0;
        }
        else{
            result[i] = 1 / ( 1 + exp( (double)(-1 * a[i])) );
        }
    }
}

void sigmoid_minus_CPU(float *a, float *result, const int size){
    for (int i = 0; i < size; i++){
        if(a[i] > 10){
            result[i] = 0;
        }
        else if(a[i] < -10){
            result[i] = 1;
        }
        else{
            result[i] = 1 / ( 1 + exp((double)a[i]) );
        }
    }
}


__global__ void sigmoid_GPU(float *a, float *result, const int size){
    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid < size){
        if(a[tid] > 10){
            result[tid] = 1;
        }
        else if(a[tid] < -10){
            result[tid] = 0;
        }
        else{
            result[tid] = 1 / ( 1 + exp( (double)(-1 * a[tid])) );
        }
    }
}

__global__ void sigmoid_minus_GPU(float *a, float *result, const int size){
    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid < size){
        if(a[tid] > 10){
            result[tid] = 0;
        }
        else if(a[tid] < -10){
            result[tid] = 1;
        }
        else{
            result[tid] = 1 / ( 1 + exp( (double)(a[tid])) );
        }
    }
}

void dz_CPU(float *y_label, float *y_pre1, float *y_pre2, float *dz, const int data_size){

        for (int i = 0; i < data_size; i++) {
            if(y_label[i] == 0){
                dz[i] = -1 * y_pre1[i];
            }
            else{
                dz[i] = -1 * y_pre2[i];
            }
        }
}

__global__ void dz_GPU(float *y_label, float *y_pre1, float *y_pre2, float *dz, float *dz1, const int data_size){

        const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (tid < data_size){
            if(y_label[tid] == 0){
                dz[tid] = y_pre1[tid];
                // dz[tid] = -50; 
            }
            else{
                dz1[tid] = -1 * y_pre2[tid];
                //dz[tid] = -50; 
            }
        }
}


void grad_CPU(float **train_data, float *dz, float *dz1, float *grad, float *grad1, 
              const int data_size, const int weight_size, const float learning_rate){
    for (int i = 0; i < data_size; i++){
        for (int j = 0; j < weight_size; j++){
            grad[j] +=  learning_rate * train_data[i][j] * dz[i];
            // grad[i] +=  learning_rate * training_x[k][i] * (predicting_z[i] - training_y[i][0]);  for multi_class
        }
    }
}

__global__ void grad_GPU(float *train_data, float *dz, float *dz1, float *grad, float *grad1, 
                    const int data_size, const int weight_size, const float learning_rate){

    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid < weight_size){
        for (int i = 0; i < data_size; i++){
            // grad[tid] +=  learning_rate * train_data[i][tid] * dz[i];
            grad[tid] +=  learning_rate * train_data[i * weight_size + tid] * dz[i];
            grad1[tid] +=  learning_rate * train_data[i * weight_size + tid] * dz1[i];
        }
        grad[tid] = grad[tid] + grad1[tid];
    }
}

__global__ void initialize_grad(float *grad, const int weight_size){
    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid < weight_size){
        grad[tid] = 0;
    }
}

void weight_update_CPU(float *weight, float *grad, const int data_size, const int weight_size){
    for(int i = 0; i < weight_size; i++){
        weight[i] -= (grad[i] / data_size);
    }
}

__global__ void weight_update_GPU(float *weight, float *grad, const int data_size, const int weight_size){
    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid < weight_size){
        grad[tid] = grad[tid] / 2556;
        weight[tid] -= grad[tid];
    }
}



int main(){
    malloc_host();
    readCSV("training_x1.csv", training_x, 2556, 1024);
    readCSV("training_y1.csv", training_y, 2556, 1);
    readCSV("testing_x1.csv", testing_x, 504, 1024);
    readCSV("testing_y1.csv", testing_y, 504, 1);
    printf("label %f\n", training_y[2036][0]);

    // for(int i = 0; i < 1024; i++){printf("h_train_data %f \n", training_x[1][i]);}

    //CPU
    float learning_rate = 0.001;

    int data_size = 2556;
    int weight_size = 1024;
    int tratin_data_bytes = 2556 * 1024 * sizeof(float);
    int weight_bytes = 1024 * sizeof(float);
    int predict_bytes = 2556 * sizeof(float);

    float *h_train_data = (float *) malloc( tratin_data_bytes ) ;
    float *h_weight = (float *) malloc( weight_bytes ) ;
    float *h_label  = (float *) malloc( predict_bytes  ) ; // host result
    float *h_predict  = (float *) malloc( predict_bytes  ) ; // host result
    float *h_softmax = (float *) malloc( predict_bytes ) ;
    float *h_softmax_minus = (float *) malloc( predict_bytes );
    float *h_dz  = (float *) malloc( predict_bytes  ) ;
    float *h_dz1  = (float *) malloc( predict_bytes  ) ;
    float *h_grad = (float *) malloc( weight_bytes ) ;
    float *h_grad1 = (float *) malloc( weight_bytes ) ;

    // Load initialize data
    for(int i = 0; i < data_size; i++){
        for(int j = 0; j < weight_size; j++){
            h_train_data[i * weight_size + j] = training_x[i][j];
        }
    }

    for(int i = 0; i < weight_size; i++){
        h_weight[i] = 0;
    }

    for(int i = 0; i < data_size; i++){
        h_label[i] = training_y[i][0];
    }

    // // GPU
    float *d_train_data, * d_weight, *d_predict, *d_softmax, *d_softmax_minus;
    float *d_label, *d_dz, *d_dz1, *d_grad, *d_grad1;

    hipGetErrorString(hipMalloc( (void **) &d_train_data, tratin_data_bytes )) ;
    hipGetErrorString(hipMalloc( (void **) &d_label, predict_bytes)) ;
    hipGetErrorString(hipMalloc( (void **) &d_weight, weight_bytes )) ;
    hipGetErrorString(hipMalloc( (void **) &d_predict, predict_bytes )) ;
    hipGetErrorString(hipMalloc( (void **) &d_softmax, predict_bytes )) ;
    hipGetErrorString(hipMalloc( (void **) &d_softmax_minus, predict_bytes )) ;
    hipGetErrorString(hipMalloc( (void **) &d_dz, predict_bytes )) ;
    hipGetErrorString(hipMalloc( (void **) &d_dz1, predict_bytes )) ;
    hipGetErrorString(hipMalloc( (void **) &d_grad, weight_bytes )) ;
    hipGetErrorString(hipMalloc( (void **) &d_grad1, weight_bytes )) ;

    hipGetErrorString(hipMemcpy( d_train_data, h_train_data, tratin_data_bytes, hipMemcpyHostToDevice )) ;
    hipGetErrorString(hipMemcpy( d_weight, h_weight, weight_bytes, hipMemcpyHostToDevice )) ;
    hipGetErrorString(hipMemcpy( d_label, h_label, predict_bytes, hipMemcpyHostToDevice ));


    // //Configure blockDim
    int bdx = 32, bdy = 32;
    while(data_size > bdx * 65535)
    {
        bdx = bdx * 2;
        bdy = bdy / 2;
    }
    while(weight_size > bdy * 65535)
    {
        bdy = bdy * 2;
        bdx = bdx / 2;
    }
    dim3 blockDim( bdx,bdy ) ; // you will want to configure this
    dim3 gridDim( (int)((data_size + blockDim.x-1)/blockDim.x), (int)((weight_size + blockDim.y-1)/blockDim.y) ) ;

    //////////////////////////////// invoke Kernel (Logistic Regression) ////////////////////////////////
 
    for(int train  = 0; train < 20; train++){
        // DOT
        Mult_GPU<<<gridDim, blockDim>>>( d_train_data, d_weight, d_predict, data_size, weight_size, 1 ) ;
        hipGetErrorString(hipDeviceSynchronize()) ;	

        //Sigmoid
        sigmoid_GPU<<<gridDim, blockDim>>>( d_predict, d_softmax, data_size ) ;
        hipGetErrorString(hipDeviceSynchronize()) ;	
        sigmoid_minus_GPU<<<gridDim, blockDim>>>( d_predict, d_softmax_minus, data_size ) ;
        hipGetErrorString(hipDeviceSynchronize()) ;	

        // Update weight (including calculating weight and )
        dz_GPU<<<gridDim, blockDim>>>(d_label, d_softmax, d_softmax_minus, d_dz, d_dz1, data_size);
        hipGetErrorString(hipDeviceSynchronize()) ;	
        grad_GPU<<<gridDim, blockDim>>>( d_train_data, d_dz, d_dz1, d_grad, d_grad1, data_size, weight_size, learning_rate);
        hipGetErrorString(hipDeviceSynchronize()) ;	
        weight_update_GPU<<<gridDim, blockDim>>>(d_weight, d_grad, data_size, weight_size);
        hipGetErrorString(hipDeviceSynchronize()) ;	
        initialize_grad<<<gridDim, blockDim>>>(d_grad, weight_size);
        hipGetErrorString(hipDeviceSynchronize()) ;
    }

    hipGetErrorString(hipMemcpy( h_weight, d_weight, weight_bytes, hipMemcpyDeviceToHost )) ;

    for(int i = 0; i < weight_size; i++){printf("h_weight %f \n", h_weight[i]);}
    //////////////////////////////// invoke Kernel (Logistic Regression) ////////////////////////////////

    // free GPU resource
    hipGetErrorString(hipFree( d_train_data )) ;
    hipGetErrorString(hipFree( d_weight )) ;
    hipGetErrorString(hipFree( d_predict )) ;
    hipGetErrorString(hipFree( d_softmax )) ;
    hipGetErrorString(hipFree( d_softmax_minus )) ;
    hipGetErrorString(hipFree( d_dz)) ;
    hipGetErrorString(hipDeviceReset()) ;

    // //Test
    float accuracy = 0;
    Mult_CPU( h_train_data, h_weight, h_predict, data_size, weight_size, 1) ;
    sigmoid_CPU( h_predict, h_softmax, data_size ) ;
    for (int i = 0; i < 2556; i++) {
        if((training_y[i][0]) == 1){
            if(h_softmax[i] > 0.5){
                accuracy++;
            }
        } 
        else{
            if(h_softmax[i] < 0.5){
                accuracy++;
            }
        }
    }
    printf("The testing accuracy is %f\n", accuracy/2556);
}
